#include "hip/hip_runtime.h"

#include <algorithm>
#include <typeinfo>

template <class T>
Buffer<T>::Buffer(int width, int height)
    : w_(width)
    , h_(height)
    , device_copy_(NULL)
{
    hipMalloc((void**)&device_data_, w_ * h_ * sizeof(T));
    host_data_ = new T[w_*h_];
    host_data_alloced_ = true;

    hipMalloc((void**)&device_copy_, sizeof(Buffer<T>));
    hipMemcpy(device_copy_, this, sizeof(Buffer<T>),
               hipMemcpyHostToDevice);
}

template <class T>
Buffer<T>::Buffer(int width, int height, T *host_data)
    : w_(width)
    , h_(height)
    , host_data_(host_data)
    , device_copy_(NULL)
{
    hipMalloc((void**)&device_data_, w_ * h_ * sizeof(T));
    host_data_alloced_ = false;

    hipMalloc((void**)&device_copy_, sizeof(Buffer<T>));
    hipMemcpy(device_copy_, this, sizeof(Buffer<T>),
               hipMemcpyHostToDevice);
}

template <class T>
Buffer<T>::~Buffer() {
    hipFree((void*)device_data_);
    hipFree((void*)device_copy_);
    if(host_data_alloced_) {
        delete [] host_data_;
    }
}

template <class T>
void Buffer<T>::clear() {
    memset(host_data_, 0, w_*h_*sizeof(T));
    hipMemset(device_data_, 0, w_*h_*sizeof(T));
}

template <class T>
void Buffer<T>::copy_device_to_host() {
    hipMemcpy(host_data_, device_data_,
               w_ * h_ * sizeof(T), hipMemcpyDeviceToHost);
}

template <class T>
void Buffer<T>::copy_host_to_device() {
    hipMemcpy(device_data_, host_data_, w_ * h_ * sizeof(T),
               hipMemcpyHostToDevice);
}

template <class T>
void Buffer<T>::copy_from_buffer(Buffer<T> *other, bool use_gpu) {
    hipMemcpy(host_data_, other->get_data(use_gpu), w_ * h_ * sizeof(T),
               hipMemcpyHostToHost);
    if(use_gpu) {
        copy_host_to_device();
    }
}

template <class T>
T* Buffer<T>::get_data(bool gpu) {
    if(gpu) {
        hipMemcpy(host_data_, device_data_,
                   w_ * h_ * sizeof(T), hipMemcpyDeviceToHost);
    }
    return host_data_;
}

template <class T>
std::size_t Buffer<T>::get_type() {
    return typeid(T).hash_code();
}

template <class T>
void Buffer<T>::set_host_data(T *new_host_data, int new_width, int new_height) {
    if(host_data_alloced_) {
        delete [] host_data_;
    }


    //Dimensions changed
    if(new_width > 0 || new_height > 0) {
        w_ = std::max(new_width, w_);
        h_ = std::max(new_height, h_);

        hipFree((void*)device_data_);
        hipMalloc((void**)&device_data_, w_ * h_ * sizeof(T));
    }
    if(new_host_data == NULL) {
        host_data_ = new T[w_*h_];
        host_data_alloced_ = true;
    }
    else {
        host_data_ = new_host_data;
        host_data_alloced_ = false;
    }
}

