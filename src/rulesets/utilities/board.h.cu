#include "hip/hip_runtime.h"

#include <algorithm>
#include <typeinfo>

template <class T>
Board<T>::Board(int width, int height)
    : width_(width)
    , height_(height)
    , device_copy_(NULL)
{
    hipMalloc((void**)&device_data_, width_ * height_ * sizeof(T));
    host_data_ = new T[width_*height_];
    host_data_alloced_ = true;

    hipMalloc((void**)&device_copy_, sizeof(Board<T>));
    hipMemcpy(device_copy_, this, sizeof(Board<T>),
               hipMemcpyHostToDevice);
}

template <class T>
Board<T>::Board(int width, int height, T *host_data)
    : width_(width)
    , height_(height)
    , host_data_(host_data)
    , device_copy_(NULL)
{
    hipMalloc((void**)&device_data_, width_ * height_ * sizeof(T));
    host_data_alloced_ = false;

    hipMalloc((void**)&device_copy_, sizeof(Board<T>));
    hipMemcpy(device_copy_, this, sizeof(Board<T>),
               hipMemcpyHostToDevice);
}

template <class T>
Board<T>::~Board() {
    hipFree((void*)device_data_);
    hipFree((void*)device_copy_);
    if(host_data_alloced_) {
        delete [] host_data_;
    }
}

template <class T>
void Board<T>::clear() {
    memset(host_data_, 0, width_*height_*sizeof(T));
    hipMemset(device_data_, 0, width_*height_*sizeof(T));
}

template <class T>
void Board<T>::copy_device_to_host() {
    hipMemcpy(host_data_, device_data_,
               width_ * height_ * sizeof(T), hipMemcpyDeviceToHost);
}

template <class T>
void Board<T>::copy_host_to_device() {
    hipMemcpy(device_data_, host_data_, width_ * height_ * sizeof(T),
               hipMemcpyHostToDevice);
}

template <class T>
void Board<T>::copy_from_board(Board<T> *other, bool use_gpu) {
    hipMemcpy(host_data_, other->get_data(use_gpu), width_ * height_ * sizeof(T),
               hipMemcpyHostToHost);
    if(use_gpu) {
        copy_host_to_device();
    }
}

template <class T>
T* Board<T>::get_data(bool gpu) {
    if(gpu) {
        hipMemcpy(host_data_, device_data_,
                   width_ * height_ * sizeof(T), hipMemcpyDeviceToHost);
    }
    return host_data_;
}

template <class T>
std::size_t Board<T>::get_type() {
    return typeid(T).hash_code();
}

template <class T>
void Board<T>::set_host_data(T *new_host_data, int new_width, int new_height) {
    if(host_data_alloced_) {
        delete [] host_data_;
    }


    //Dimensions changed
    if(new_width > 0 || new_height > 0) {
        width_ = std::max(new_width, width_);
        height_ = std::max(new_height, height_);

        hipFree((void*)device_data_);
        hipMalloc((void**)&device_data_, width_ * height_ * sizeof(T));
    }
    if(new_host_data == NULL) {
        host_data_ = new T[width_*height_];
        host_data_alloced_ = true;
    }
    else {
        host_data_ = new_host_data;
        host_data_alloced_ = false;
    }
}

