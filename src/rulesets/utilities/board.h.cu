
#include <typeinfo>

template <class T>
Board<T>::Board(int width, int height)
    : width_(width)
    , height_(height)
{
    hipMalloc((void**)&device_data_, width_ * height_ * sizeof(T));
    host_data_ = new T[width_*height_];

    hipMalloc((void**)&device_copy_, sizeof(Board<T>));
    hipMemcpy(device_copy_, this, sizeof(Board<T>),
               hipMemcpyHostToDevice);
}

template <class T>
Board<T>::~Board() {
    hipFree((void*)device_data_);
    hipFree((void*)device_copy_);
    delete [] host_data_;
}

template <class T>
void Board<T>::clear() {
    memset(host_data_, 0, width_*height_*sizeof(T));
    hipMemset(device_data_, 0, width_*height_*sizeof(T));
}

template <class T>
void Board<T>::copy_device_to_host() {
    hipMemcpy(host_data_, device_data_,
               width_ * height_ * sizeof(T), hipMemcpyDeviceToHost);
}

template <class T>
void Board<T>::copy_host_to_device() {
    hipMemcpy(device_data_, host_data_, width_ * height_ * sizeof(T),
               hipMemcpyHostToDevice);
}

template <class T>
T* Board<T>::get_data(bool gpu) {
    if(gpu) {
        hipMemcpy(host_data_, device_data_,
                   width_ * height_ * sizeof(T), hipMemcpyDeviceToHost);
    }
    return host_data_;
}

template <class T>
std::size_t Board<T>::get_type() {
    return typeid(T).hash_code();
}

