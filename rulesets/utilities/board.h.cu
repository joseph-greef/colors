
template <class T>
Board<T>::Board(int width, int height, bool gpu)
    : width_(width)
    , height_(height)
    , gpu_(gpu)
{
    if(gpu_) {
        hipMalloc((void**)&data_, width_ * height_ * sizeof(T));
    }
    else {
        data_ = new T[width_*height_];
    }
}

template <class T>
Board<T>::~Board() {
    if(gpu_) {
        hipFree((void*)data_);
    }
    else {
        delete [] data_;
    }
}

template <class T>
void Board<T>::copy_board_from(Board<T> &other) {
    if(gpu_ && other.gpu_) {
        hipMemcpy(data_, other.data_, width_ * height_ * sizeof(T),
                   hipMemcpyDeviceToDevice);
    }
    else if(!gpu_ && other.gpu_) {
        hipMemcpy(data_, other.data_, width_ * height_ * sizeof(T),
                   hipMemcpyDeviceToHost);
    }
    else if(gpu_ && !other.gpu_) {
        hipMemcpy(data_, other.data_, width_ * height_ * sizeof(T),
                   hipMemcpyHostToDevice);
    }
    else if(!gpu_ && !other.gpu_) {
        hipMemcpy(data_, other.data_, width_ * height_ * sizeof(T),
                   hipMemcpyHostToHost);
    }
}

